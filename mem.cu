#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void ikernel()
{
    printf("hello world\n");
}

int main()
{
    // cuda hello world
    hipStream_t stream;
    hipStreamCreate(&stream);
    ikernel<<<4, 4, 0, stream>>>();
    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);
    return 0;
}